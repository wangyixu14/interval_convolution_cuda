#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <cstdlib>
#include <time.h>
#include <cutil.h>
#include <iostream>
#include <assert.h>

// the input matrix size
#define WA 512 
#define HA 512    

#define POOL_SIZE 3
#define TILE_SIZE 14
#define BLOCK_SIZE (TILE_SIZE + POOL_SIZE - 1)

/*cuda version of max pooling */
__global__ void Max_Pooling(float* In_lower, float* In_upper, float* Out_lower, float* Out_upper)
{
    __shared__ float tile_lower[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float tile_upper[BLOCK_SIZE][BLOCK_SIZE];

    // get thread indices
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // get the output indices
    int row_o = ty + blockIdx.y * TILE_SIZE;
    int col_o = tx + blockIdx.x * TILE_SIZE;

    // shift to obtain input indices
    int row_i = row_o - POOL_SIZE / 2;
    int col_i = col_o - POOL_SIZE / 2;

    // Load tile elements
    if(row_i >= 0 && row_i < HA && col_i >= 0 && col_i < WA){
        tile_lower[ty][tx] = In_lower[row_i * WA + col_i];
        tile_upper[ty][tx] = In_upper[row_i * WA + col_i];
    }
    else{
        tile_lower[ty][tx] = -(float)RAND_MAX;
        tile_upper[ty][tx] = -(float)RAND_MAX;
    }

    __syncthreads();

    if(tx < TILE_SIZE && ty < TILE_SIZE){
        float Value_lower = -(float)RAND_MAX;
        float Value_upper = -(float)RAND_MAX;
        
        for(int y = 0; y < POOL_SIZE; y++)
            for(int x = 0; x < POOL_SIZE; x++){
            	if (tile_upper[y + ty][x + tx] >= Value_upper){
            		Value_upper = tile_upper[y + ty][x + tx];
            		Value_lower = tile_lower[y + ty][x + tx];
            	}
            }
        if(row_o < HA && col_o < WA){
            Out_lower[row_o * WA + col_o] = Value_lower;
            Out_upper[row_o * WA + col_o] = Value_upper;
        }
    }
}


/* cpu version of max pooling */
void reference(float* in_lower, float* in_upper, float* out_lower, float* out_upper, int size_X, int size_Y, int pool_size)
{
    int p_cent = (pool_size - 1) / 2;
	for(int y = 0; y < size_Y; y++){ 
		for(int x = 0; x < size_X; x++){
			float value_upper = -(float)RAND_MAX;
			float value_lower = -(float)RAND_MAX;

			for(int j = -p_cent; j <= p_cent; j++){ 
				for(int i = -p_cent; i <= p_cent; i++){ 
					if (x + i > -1 && x + i < size_X && y + j > -1 && y + j < size_Y){
						if (in_upper[(x + i) + (y + j) * size_X] >= value_upper){
							value_upper = in_upper[(x + i) + (y + j) * size_X];
							value_lower = in_lower[(x + i) + (y + j) * size_X];
						}
					}
				}
			}

			out_lower[x + y * size_X] = value_lower;
			out_upper[x + y * size_X] = value_upper;
		}  		
	}
}

void InitInterval(float* data_lower, float* data_upper, int size)
{
	for (int i = 0; i < size; ++i){
		int v_1 = rand() / (float)RAND_MAX;
		int v_2 = rand() / (float)RAND_MAX;
		if (v_1 >= v_2){
			data_lower[i] = v_2;
			data_upper[i] = v_1;
		}
		else{
			data_lower[i] = v_1;
			data_upper[i] = v_2;
		}
	}
}

int main(int argc, char** argv)
{
	srand(2006);

	hipEvent_t start_G, stop_G;

	hipEventCreate(&start_G);
	hipEventCreate(&stop_G);

	unsigned int size_A = WA * HA;
	unsigned int mem_size_A = sizeof(float) * size_A;
	float* h_A_1 = (float*)malloc(mem_size_A);
	float* h_A_2 = (float*)malloc(mem_size_A);
	float* h_B_1 = (float*)malloc(mem_size_A);
	float* h_B_2 = (float*)malloc(mem_size_A);
	float* h_cpu_1 = (float*)malloc(mem_size_A);
	float* h_cpu_2 = (float*)malloc(mem_size_A);

	InitInterval(h_A_1, h_A_2, size_A);

    unsigned int timer;
    CUT_SAFE_CALL(cutCreateTimer(&timer));
	cutStartTimer(timer);
    reference(h_A_1, h_A_2, h_cpu_1, h_cpu_2, WA, HA, POOL_SIZE);
	cutStopTimer(timer);
    printf("Host CPU Processing time: %f (ms)\n", cutGetTimerValue(timer));
	
	float* d_A_1;
	float* d_A_2;
	float* d_B_1;
	float* d_B_2;

	hipMalloc((void**)&d_A_1, mem_size_A);
	hipMalloc((void**)&d_A_2, mem_size_A);
	hipMalloc((void**)&d_B_1, mem_size_A);
	hipMalloc((void**)&d_B_2, mem_size_A);

	hipMemcpy(d_A_1, h_A_1, mem_size_A, hipMemcpyHostToDevice);
	hipMemcpy(d_A_2, h_A_2, mem_size_A, hipMemcpyHostToDevice);
	hipMemcpy(d_B_1, h_B_1, mem_size_A, hipMemcpyHostToDevice);
	hipMemcpy(d_B_1, h_B_2, mem_size_A, hipMemcpyHostToDevice);

	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid((WA - 1) / (BLOCK_SIZE - POOL_SIZE + 1) + 1, (WA - 1) / (BLOCK_SIZE - POOL_SIZE + 1) + 1);

	hipEventRecord(start_G);
	Max_Pooling <<< grid, threads >>>(d_A_1, d_A_2, d_B_1, d_B_2);

	hipDeviceSynchronize();
	hipEventRecord(stop_G);
	hipEventSynchronize(stop_G);

	hipMemcpy(h_B_1, d_B_1, mem_size_A, hipMemcpyDeviceToHost);
	hipMemcpy(h_B_2, d_B_2, mem_size_A, hipMemcpyDeviceToHost);

	float miliseconds = 0;
	hipEventElapsedTime(&miliseconds, start_G, stop_G);

	printf("Time took to compute matrix A of dimensions %d x %d  on GPU is %f ms \n \n \n", WA, HA, miliseconds);

	unsigned int result_regtest_1 = cutComparefe(h_cpu_1, h_B_1, WA*HA, 0.001f);
	unsigned int result_regtest_2 = cutComparefe(h_cpu_2, h_B_2, WA*HA, 0.001f);
    printf( "Test %s\n", (result_regtest_1 == 1 && result_regtest_2 == 1) ? "PASSED" : "FAILED");

	free(h_A_1);
	free(h_A_2);
	free(h_B_1);
	free(h_B_2);

	hipFree(d_A_1);
	hipFree(d_A_2);
	hipFree(d_B_1);
	hipFree(d_B_2);

	return EXIT_SUCCESS;
}