#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and 
 * international Copyright laws.  
 *
 * This software and the information contained herein is PROPRIETARY and 
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and 
 * conditions of a Non-Disclosure Agreement.  Any reproduction or 
 * disclosure to any third party without the express written consent of 
 * NVIDIA is prohibited.     
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH 
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.   
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE 
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.  This source code is a "commercial item" as 
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of 
 * "commercial computer software" and "commercial computer software 
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995) 
 * and is provided to the U.S. Government only as a commercial end item.  
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through 
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

/* Matrix multiplication: C = A * B.
 * Device code.
 */

#ifndef _MATRIXMUL_KERNEL_H_
#define _MATRIXMUL_KERNEL_H_

#include <stdio.h>
#include "matrixmul.h"
#define TILE_WIDTH 32

////////////////////////////////////////////////////////////////////////////////
//! Simple test kernel for device functionality
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
// Matrix multiplication kernel thread specification
__global__ void MatrixMulKernel(Matrix M, Matrix N, Matrix P)
{
	__shared__ float tileMs[TILE_WIDTH][TILE_WIDTH + 1];
	__shared__ float tileNs[TILE_WIDTH][TILE_WIDTH + 1];

	int tx = threadIdx.x; int ty = threadIdx.y;
	int bx = blockIdx.x; int by = blockIdx.y;

	int row = by * TILE_WIDTH + ty;
	int col = bx * TILE_WIDTH + tx;

	float V = 0;

	for(int i = 0; i < ceilf(M.width / (float)TILE_WIDTH); i++){
		if(row < M.height && (i * TILE_WIDTH + tx) < M.width)
			tileMs[ty][tx] = M.elements[row * M.width + i * TILE_WIDTH + tx];
        else
            tileMs[ty][tx] = 0;
		if(col < N.width && (i * TILE_WIDTH + ty) < N.height)
			tileNs[ty][tx] = N.elements[(i * TILE_WIDTH + ty) * N.width + col];
        else
            tileNs[ty][tx] = 0;
		__syncthreads();

		for(int j = 0; j < TILE_WIDTH; j++)
			V += tileMs[ty][j] * tileNs[j][tx];
		
		__syncthreads();
	}
	if(row < P.height && col < P.width)
		P.elements[row * P.width + col] = V;

}

#endif // #ifndef _MATRIXMUL_KERNEL_H_
